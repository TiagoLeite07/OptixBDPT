#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "random.h"

rtDeclareVariable(float4, Kd, , );
rtDeclareVariable(float4, Ks, , );
rtDeclareVariable(float, shininess, , );


// Intersection attributes
rtDeclareVariable(float3, pos, attribute pos, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
//rtDeclareVariable(float3, tangent, attribute tangent, );
//rtDeclareVariable(float3, bitangent, attribute bitangent, );

struct PathResult{
	float4 result;
	float4 atenuation;
	float3 position;
	float3 direction;
	unsigned int depth;
	unsigned int seed;
	bool count_emissive;
	bool finished;
};

rtDeclareVariable(PathResult, current_path_result, rtPayload, );
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, );

__device__ __inline__ void calc_direct_light(){

}

RT_PROGRAM void closest_hit(){
	//because we calculate direct lighting in every point of the path,
	//when first diffuse material is hit we stop counting emmisive contributions
	current_path_result.count_emissive=false;
	//calculate diffuse and specular probabilities.
	float pdiff=(Kd.x+Kd.y+Kd.z)*0.33333333333333333333333333333f;
	float pspec=(Ks.x+Ks.y+Ks.z)*0.33333333333333333333333333333f;
	pspec*=fminf(1.f, optix::dot(current_ray.direction, shading_normal)*(shininess+2.f)/(shininess+1.f));

	//randomly select the type of contribution
	float r=rnd(current_path_result.seed);
	if(r<pdiff+pspec){
		//select diffuse sample
		if(r<pdiff){
			float u1=rnd(current_path_result.seed);
			float u2=rnd(current_path_result.seed);
			float3 dir;
			optix::cosine_sample_hemisphere(u1, u2, dir);
			optix::Onb onb(shading_normal);
			onb.inverse_transform(dir);

			current_path_result.atenuation *= Kd/pdiff;
			current_path_result.direction = dir;

		}
		//select specular sample
		else {
			float u1=rnd(current_path_result.seed);
			float u2=rnd(current_path_result.seed);
			float3 dir;
			dir.x = sqrtf(1-powf(u1, 2.f/(shininess+1.f)))*cosf(M_2_PIf*u2);
			dir.y = sqrtf(1-powf(u1, 2.f/(shininess+1.f)))*sinf(M_2_PIf*u2);
			dir.z = powf(u1, 1.f/(shininess+1.f));
			optix::Onb onb(optix::reflect(current_ray.direction, shading_normal));
			onb.inverse_transform(dir);

			float intensity=optix::dot(dir, shading_normal);
			//verify if sampled direction is above surface
			if(intensity>0.f){
				current_path_result.atenuation*= ((shininess+2.f)/(shininess+1.f)) * (Ks/pspec) * optix::dot(dir, shading_normal);
				current_path_result.direction=dir;
			}
			else{
				current_path_result.finished=true;
			}
		}
	}
	//consider that photon is absorbed and finish path
	else{
		current_path_result.finished=true;
	}

}
