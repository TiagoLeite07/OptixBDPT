#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "random.h"

rtDeclareVariable(float4, Kd, , );
rtDeclareVariable(float4, Ks, , );
rtDeclareVariable(float, shininess, , );


// Intersection attributes
rtDeclareVariable(float3, pos, attribute pos, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
//rtDeclareVariable(float3, tangent, attribute tangent, );
//rtDeclareVariable(float3, bitangent, attribute bitangent, );

struct PathResult{
	float4 result;
	float4 atenuation;
	float3 position;
	float3 direction;
	unsigned int seed;
	bool count_emissive;
	bool finished;
};

rtDeclareVariable(PathResult, current_path_result, rtPayload, );
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, );

__device__ __inline__ void calc_direct_light(){

}

RT_PROGRAM void closest_hit(){
	current_path_result.count_emissive=false;
	float pdiff=(Kd.x+Kd.y+Kd.z)*0.33333333333333333333333333333f;
	float pspec=(Ks.x+Ks.y+Ks.z)*0.33333333333333333333333333333f;

	float r=rnd(current_path_result.seed);

	if(r<pdiff+pspec){

		if(r<pdiff){
			float u1=rnd(current_path_result.seed);
			float u2=rnd(current_path_result.seed);
			float3 dir;
			optix::cosine_sample_hemisphere(u1, u2, dir);
			optix::Onb onb(shading_normal);
			onb.inverse_transform(dir);

			current_path_result.atenuation *= Kd/pdiff;
			current_path_result.direction = dir;

		}
		else {
			float u1=rnd(current_path_result.seed);
			float u2=rnd(current_path_result.seed);
			float3 dir;
			dir.x = sqrtf(1-powf(u1, 2.f/(shininess+1.f)))*cosf(M_2_PIf*u2);
			dir.y = sqrtf(1-powf(u1, 2.f/(shininess+1.f)))*sinf(M_2_PIf*u2);
			dir.z = powf(u1, 1.f/(shininess+1.f));
			optix::Onb onb(optix::reflect(current_ray.direction, shading_normal));
			onb.inverse_transform(dir);

			float intensity=dot(dir, shading_normal);

			if(intensity>0.f){
				current_path_result.atenuation*= ((shininess+2.f)/(shininess+1.f)) * (Ks/pspec) * optix::dot(dir, shading_normal);
				current_path_result.direction=dir;
			}
			else{
				current_path_result.finished=true;
			}
		}
	}
	else{
		current_path_result.finished=true;
	}

}
