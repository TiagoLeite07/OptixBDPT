#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "sphere_light.h"

using namespace optix;

rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, );

RT_PROGRAM void sphere_light_bounding_box(int primIdx, float result[6]){
	Aabb *bb = (Aabb*) result;
	float3 center = make_float3(lights[primIdx].pos);
	float3 rad = make_float3(lights[primIdx].pos.w);
	float3 min = center-rad;
	float3 max = center+rad;

	bb->m_min=min;
	bb->m_max=max;
}

RT_PROGRAM void sphere_light_intersect(int primIdx){
	float3 center = make_float3(lights[primIdx].pos);
	float radius = lights[primIdx].pos.w;

	float3 o = current_ray.origin - center;
	float b = dot(o, current_ray.direction);
	float c = dot(o, o) - radius * radius;
	float disc = b * b - c;
	if(disc>0.f){
		float sdisc = sqrtf(disc);
		float root1 = (-b -sdisc);

		if(rtPotentialIntersection(root1)){
			//TODO define attributes
			light = lights[primIdx];
			if(rtReportIntersection(0))
				return;
		}
		float root2 = (-b + sdisc);
		if(rtPotentialIntersection(root2)){
			light = lights[primIdx];
			rtReportIntersection(0);
		}
	}
}
