#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "mesh.h"

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, );

RT_PROGRAM void intersectMesh(int primIdx){
	//get indices
	int3 id=index_buffer[primIdx];
	//get vertices
	float3 v1=vertex_buffer[id.x];
	float3 v2=vertex_buffer[id.y];
	float3 v3=vertex_buffer[id.z];
	//intersect ray with triangle
	float3 n;
	float t, beta, gamma;
	if(optix::intersect_triangle(current_ray, v1, v2, v3, n, t, beta, gamma))
	{
		if(rtPotentialIntersection(t))
		{
			//loading normals
			float3 n1=normal_buffer[id.x];
			float3 n2=normal_buffer[id.y];
			float3 n3=normal_buffer[id.z];

			if(tangent_buffer.size()==vertex_buffer.size()){
				float3 t1=tangent_buffer[id.x];
				float3 t2=tangent_buffer[id.y];
				float3 t3=tangent_buffer[id.z];

				float3 b1=bitangent_buffer[id.x];
				float3 b2=bitangent_buffer[id.y];
				float3 b3=bitangent_buffer[id.z];

				tangent=(1.0f-beta-gamma)*t1 + beta*t2 +gamma*t3;
				bitangent=(1.0f-beta-gamma)*b1 + beta*b2 +gamma*b3;
			}
			else{
				tangent=make_float3(0.f);
				bitangent=make_float3(0.f);
			}


			//loading texCoords
			if(texCoord_buffer.size()==vertex_buffer.size()){
				float2 t1=texCoord_buffer[id.x];
				float2 t2=texCoord_buffer[id.y];
				float2 t3=texCoord_buffer[id.z];
				texCoord=(1.0f-beta-gamma)*t1 + beta*t2 +gamma*t3;
			}
			else
			{
				texCoord=make_float2(1.0f,0.0f);
			}
			//setting attributes
			shading_normal=(1.0f-beta-gamma)*n1 + beta*n2 +gamma*n3;
			geometric_normal=optix::normalize(n);
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void boundingBoxMesh(int primIdx, float result[6]){
    //get indices
    int3 id=index_buffer[primIdx];
    //load vertices
    float3 v1=vertex_buffer[id.x];
    float3 v2=vertex_buffer[id.y];
    float3 v3=vertex_buffer[id.z];
    const float area = optix::length(optix::cross(v2-v1,v3-v1));
    optix::Aabb* aabb = (optix::Aabb*)result;
    if(area>0.0f)
    {
        aabb->m_min=fminf(fminf(v1,v2),v3);
        aabb->m_max=fmaxf(fmaxf(v1,v2),v3);
    }
    else
    {
        aabb->invalidate();
    }
}
