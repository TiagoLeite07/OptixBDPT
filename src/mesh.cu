#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <hip/hip_math_constants.h>
#include "mesh.h"

#define BUMP_INTENSITY 0.1f

#define TESS 4

using namespace optix;

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, );

rtTextureSampler<float, 2> bump;

RT_PROGRAM void intersectMesh(int primIdx){
	//get indices
	int3 id=index_buffer[primIdx];
	//get vertices
	float3 v1=vertex_buffer[id.x];
	float3 v2=vertex_buffer[id.y];
	float3 v3=vertex_buffer[id.z];
	//intersect ray with triangle
	float3 n;
	float t, beta, gamma;
	if(intersect_triangle(current_ray, v1, v2, v3, n, t, beta, gamma))
	{
		//loading normals
		float3 n1=normal_buffer[id.x];
		float3 n2=normal_buffer[id.y];
		float3 n3=normal_buffer[id.z];

		float3 tmp_normal = (1.0f-beta-gamma)*n1 + beta*n2 +gamma*n3;


		//loading texCoords
		float2 tmp_texCoord;
		if(texCoord_buffer.size()==vertex_buffer.size()){
			float2 t1=texCoord_buffer[id.x];
			float2 t2=texCoord_buffer[id.y];
			float2 t3=texCoord_buffer[id.z];
			tmp_texCoord=(1.0f-beta-gamma)*t1 + beta*t2 +gamma*t3;
		}
		else
		{
			tmp_texCoord=make_float2(1.0f,0.0f);
		}

		float3 tmp_tangent, tmp_bitangent;

		if(tangent_buffer.size()==vertex_buffer.size()){
			float3 t1=tangent_buffer[id.x];
			float3 t2=tangent_buffer[id.y];
			float3 t3=tangent_buffer[id.z];

			float3 b1=bitangent_buffer[id.x];
			float3 b2=bitangent_buffer[id.y];
			float3 b3=bitangent_buffer[id.z];



			tmp_tangent=(1.0f-beta-gamma)*t1 + beta*t2 +gamma*t3;
			tmp_bitangent=(1.0f-beta-gamma)*b1 + beta*b2 +gamma*b3;


		}
		else{
			tmp_tangent=make_float3(0.f);
			tmp_bitangent=make_float3(0.f);
		}

		if((tangent_buffer.size()==vertex_buffer.size()) && (texCoord_buffer.size()==vertex_buffer.size())){

			//bump mapping
			float height = tex2D(bump, texCoord.x, texCoord.y);

			float dx = (tex2D(bump, tmp_texCoord.x+0.0005f, tmp_texCoord.y) - tex2D(bump, tmp_texCoord.x-0.0005f, tmp_texCoord.y))*1000.f;
			float dy = (tex2D(bump, tmp_texCoord.x, tmp_texCoord.y+0.0005f) - tex2D(bump, tmp_texCoord.x, tmp_texCoord.y-0.0005f))*1000.f;
			if(dx>0.f || dy >0.f){
				float3 o = normalize(-dx*tmp_tangent + dy*tmp_bitangent - sqrtf(dx*dx + dy*dy) * tmp_normal);
				float3 s = normalize(cross(tmp_normal, o));
				tmp_normal= normalize(cross(o,s));

			}

			//parallax mapping
			/*Matrix3x3 a;
			a.setCol(0, tmp_tangent);
			a.setCol(1, tmp_bitangent);
			a.setCol(2, tmp_normal);

			float3 eye_vec = (-current_ray.direction) * a;

			float parallax_limit = -length(make_float2(eye_vec))/eye_vec.z;
			parallax_limit*=BUMP_INTENSITY;

			float2 offSetDir=normalize(make_float2(eye_vec));
			float2 maxOffset = offSetDir * parallax_limit;

			int nSamples = (int) optix::lerp(1000.f, 500.f, dot(-current_ray.direction, tmp_normal));
			float step = 1.f/(float) nSamples;

			float ray_h = 1.f;
			float2 current_offset=make_float2(0.f);
			float2 last_offset=make_float2(0.f);

			float last_h=1.f;
			float current_h=1.f;

			int current_sample=0;

			while(current_sample<nSamples){
				current_h = tex2D(bump, tmp_texCoord.x + current_offset.x, tmp_texCoord.y - current_offset.y);
				if(current_h > ray_h){
					float delta1 = current_h - ray_h;
					float delta2 = (ray_h + step) - last_h;

					float ratio = delta1 / (delta1 + delta2);

					current_offset = ratio * last_offset + (1.f-ratio) * current_offset;

					current_sample = nSamples+1;
				}
				else{
					current_sample++;

					ray_h -= step;

					last_offset = current_offset;
					current_offset += step * maxOffset;

					last_h = current_h;
				}
			}

			tmp_texCoord.x += current_offset.x;
			tmp_texCoord.y -= current_offset.y;

			//t += current_h * dot(tmp_normal, -current_ray.direction);

			if(t< 0.1) return;

			float dx = (tex2D(bump, tmp_texCoord.x+0.0005f, tmp_texCoord.y) - tex2D(bump, tmp_texCoord.x-0.0005f, tmp_texCoord.y))*1000.f;
			float dy = (tex2D(bump, tmp_texCoord.x, tmp_texCoord.y+0.0005f) - tex2D(bump, tmp_texCoord.x, tmp_texCoord.y-0.0005f))*1000.f;
			if(dx>0.f || dy >0.f){
				float3 o = normalize(-dx*tmp_tangent + dy*tmp_bitangent - sqrtf(dx*dx + dy*dy) * tmp_normal);
				float3 s = normalize(cross(tmp_normal, o));
				tmp_normal= cross(o,s);

			}*/
		}

		if(rtPotentialIntersection(t))
		{

			//setting attributes
			shading_normal=tmp_normal;
			geometric_normal=optix::normalize(n);
			texCoord=tmp_texCoord;
			tangent=tmp_tangent;
			bitangent=tmp_bitangent;
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void boundingBoxMesh(int primIdx, float result[6]){
    //get indices
    int3 id=index_buffer[primIdx];
    //load vertices
    float3 v1=vertex_buffer[id.x];
    float3 v2=vertex_buffer[id.y];
    float3 v3=vertex_buffer[id.z];

    /*float3 v1d=vertex_buffer[id.x]-normal_buffer[id.x]* 1.5f * BUMP_INTENSITY;
    float3 v2d=vertex_buffer[id.y]-normal_buffer[id.y]* 1.5f * BUMP_INTENSITY;
    float3 v3d=vertex_buffer[id.z]-normal_buffer[id.z]* 1.5f * BUMP_INTENSITY;*/
    const float area = optix::length(optix::cross(v2-v1,v3-v1));
    Aabb* aabb = (optix::Aabb*)result;
    if(area>0.0f)
    {
        /*aabb->m_min=fminf(fminf(fminf(v1, v1d),fminf(v2, v2d)), fminf(v3, v3d));
        aabb->m_max=fmaxf(fmaxf(fmaxf(v1, v1d),fmaxf(v2, v2d)), fmaxf(v3, v3d));*/

        aabb->m_min=fminf(fminf(v1,v2), v3);
        aabb->m_max=fmaxf(fmaxf(v1,v2), v3);
    }
    else
    {
        aabb->invalidate();
    }
}
