#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "commonStructs.h"
#include "random.h"
#include "sphere_light.h"

using namespace optix;

struct PathResult{
	float4 result;
	float4 atenuation;
	float3 position;
	float3 direction;
	unsigned int depth;
	unsigned int seed;
	bool count_emissive;
	bool finished;
};

struct ShadowResult{
	bool in_shadow;
};

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, );

//ray payloads
rtDeclareVariable(PathResult, current_path_result, rtPayload, );
rtDeclareVariable(ShadowResult, current_shadow_result, rtPayload, );

//kernel dimensions
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

//camera properties
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
//lens (for depth of field)
rtDeclareVariable(float, lens_radius, , );
rtDeclareVariable(float, focal_dist, , );

//frame number to make sure result is different every frame
rtDeclareVariable(int, frame, , );
//samples for stratified sampling
rtDeclareVariable(int, sqrt_num_samples, , )=2;

//output buffer
rtBuffer<float4, 2> output;

//top object to start tracing rays
rtDeclareVariable(rtObject, top_object, , );

RT_PROGRAM void camera(){
	unsigned int seed = tea<16>(launch_dim.x*launch_index.y+launch_index.x, frame);

	float2 inv_screen=1.f/(make_float2(launch_dim)) *2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;
	float2 jitter_scale = inv_screen / sqrt_num_samples;

	int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;

	float4 result = make_float4(0.0f);

	do{
		int x = samples_per_pixel%sqrt_num_samples;
		int y = samples_per_pixel/sqrt_num_samples;
		float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
		float2 d = pixel + jitter*jitter_scale;
		float3 ray_origin = eye;
		float3 ray_direction = normalize(d.x * U + d.y * V + W);

		PathResult ray_result;
		ray_result.atenuation=make_float4(1.f);
		ray_result.count_emissive=true;
		ray_result.depth=0;
		ray_result.result=make_float4(0.f);
		ray_result.seed=seed;
		ray_result.finished=false;

		for(;;){

			Ray ray = optix::make_Ray(ray_origin, ray_direction, PathRay, 0.001, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, ray_result);

			if(ray_result.finished)
				break;

			ray_origin=ray_result.position;
			ray_direction=ray_result.direction;

		}

		result+=ray_result.result;
		seed=ray_result.seed;

	}while(--samples_per_pixel);

	result/=sqrt_num_samples * sqrt_num_samples;

	if(frame>1){
		float a = 1.f/float(frame);
		float b = float(frame-1)*a;
		float4 old_color=output[launch_index];
		output[launch_index]=a*result+b*old_color;
	}
	else{
		output[launch_index]=result;
	}

}

RT_PROGRAM void exception(){
	rtPrintExceptionDetails();
}

RT_PROGRAM void path_miss(){
	current_path_result.finished = true;
}

#include "material.h"

__device__ __inline__ void calc_direct_light(){

}

RT_PROGRAM void glossy_shading(){
	//because we calculate direct lighting in every point of the path,
	//when first diffuse material is hit we stop counting emmisive contributions
	current_path_result.count_emissive=false;
	//calculate diffuse and specular probabilities.
	float pdiff=(Kd.x+Kd.y+Kd.z)*0.33333333333333333333333333333f;
	float pspec=(Ks.x+Ks.y+Ks.z)*0.33333333333333333333333333333f;
	pspec*=fminf(1.f, optix::dot(current_ray.direction, shading_normal)*(Ns+2.f)/(Ns+1.f));

	//randomly select the type of contribution
	float r=rnd(current_path_result.seed);
	if(r<pdiff+pspec){
		//select diffuse sample
		if(r<pdiff){
			float u1=rnd(current_path_result.seed);
			float u2=rnd(current_path_result.seed);
			float3 dir;
			optix::cosine_sample_hemisphere(u1, u2, dir);
			optix::Onb onb(shading_normal);
			onb.inverse_transform(dir);

			current_path_result.atenuation *= Kd/pdiff;
			current_path_result.direction = dir;

		}
		//select specular sample
		else {
			float u1=rnd(current_path_result.seed);
			float u2=rnd(current_path_result.seed);
			float3 dir;
			dir.x = sqrtf(1-powf(u1, 2.f/(Ns+1.f)))*cosf(M_2_PIf*u2);
			dir.y = sqrtf(1-powf(u1, 2.f/(Ns+1.f)))*sinf(M_2_PIf*u2);
			dir.z = powf(u1, 1.f/(Ns+1.f));
			optix::Onb onb(optix::reflect(current_ray.direction, shading_normal));
			onb.inverse_transform(dir);

			float intensity=optix::dot(dir, shading_normal);
			//verify if sampled direction is above surface
			if(intensity>0.f){
				current_path_result.atenuation*= ((Ns+2.f)/(Ns+1.f)) * (Ks/pspec) * optix::dot(dir, shading_normal);
				current_path_result.direction=dir;
			}
			else{
				current_path_result.finished=true;
			}
		}
	}
	//consider that photon is absorbed and finish path
	else{
		current_path_result.finished=true;
	}

}

RT_PROGRAM void path_ignore_alpha(){
	float4 color=Kd*tex2D(map_Kd, texCoord.x, texCoord.y);
	if(color.w == 0.f) rtIgnoreIntersection();
}


RT_PROGRAM void shadow_probe(){
	float4 color=Kd*tex2D(map_Kd, texCoord.x, texCoord.y);
	if(color.w == 0.f) rtIgnoreIntersection();
	else{
		current_shadow_result.in_shadow=true;
		rtTerminateRay();
	}
}




