#include "hip/hip_runtime.h"
#include <optix_world.h>

#include "random.h"
#include "sphere_light.h"

#include "commonStructs.h"

using namespace optix;



struct PathResult{
	float4 result;
	float4 atenuation;
	float3 position;
	float3 direction;
	float weight;
	unsigned int depth;
	unsigned int seed;
	bool count_emissive;
	bool finished;
};

struct ShadowResult{
	bool in_shadow;
};

#define MIN_DEPTH 3

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, );

//ray payloads
rtDeclareVariable(PathResult, current_path_result, rtPayload, );
rtDeclareVariable(ShadowResult, current_shadow_result, rtPayload, );
rtDeclareVariable(LightPathResult, current_light_result, rtPayload, );

//kernel dimensions
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

//camera properties
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
//lens (for depth of field)
rtDeclareVariable(float, lens_radius, , )=0.0f;
rtDeclareVariable(float, focal_dist, , )=150.f;

//frame number to make sure result is different every frame
rtDeclareVariable(int, frame, , );
//samples for stratified sampling
rtDeclareVariable(int, sqrt_num_samples, , )=1;

//light path buffer
rtBuffer<LightPathResult, 3> lightPathBuffer;

//output buffer
rtBuffer<float4, 2> output;

//top object to start tracing rays
rtDeclareVariable(rtObject, top_object, , );


rtDeclareVariable(float, scene_epsilon, , )=0.01f;


RT_PROGRAM void light_path_gen(){
	unsigned int seed = tea<16>(launch_dim.x*launch_index.y+launch_index.x, frame);
	//TODO calculate light path
	SphereLight l = lights[0];
	float l1 = rnd(seed)*2.f-1.f;
	float l2 = rnd(seed)*2.f-1.f;
	while((l1*l1+l2*l2)>=1.f){
		l1 = rnd(seed)*2.f-1.f;
		l2 = rnd(seed)*2.f-1.f;
	}

	float3 light_normal;
	light_normal.x = 2.f * l1 * sqrtf(1.f - l1*l1 - l2*l2);
	light_normal.y = 2.f * l2 * sqrtf(1.f - l1*l1 - l2*l2);
	light_normal.z = 1.f - 2.f * (l1*l1 + l2*l2);

	float3 light_point = make_float3(l.pos) + l.pos.w * light_normal;

	l1 = rnd(seed);
	l2 = rnd(seed);

	float3 light_dir;
	optix::cosine_sample_hemisphere(l1, l2, light_dir);
	optix::Onb onb_light(light_normal);
	onb_light.inverse_transform(light_dir);

	Ray light_ray = optix::make_Ray(light_point, light_dir, LightPathRay, scene_epsilon, RT_DEFAULT_MAX);

	LightPathResult result0;
	result0.radiance=l.color;
	result0.In=light_dir;

	rtTrace(top_object, light_ray, result0);
	lightPathBuffer[make_uint3(launch_index, 0)]=result0;

	int i=1;

	while((i < LIGHT_PATH_LENGTH) && (!lightPathBuffer[make_uint3(launch_index, i-1)].missed)){


		float4 diff_coef = lightPathBuffer[make_uint3(launch_index, i-1)].Kd;
		float4 spec_coef = lightPathBuffer[make_uint3(launch_index, i-1)].Ks;

		float3 position = lightPathBuffer[make_uint3(launch_index, i-1)].position;

		float3 ffnormal = optix::faceforward(lightPathBuffer[make_uint3(launch_index, i-1)].normal, -lightPathBuffer[make_uint3(launch_index, i-1)].In, lightPathBuffer[make_uint3(launch_index, i-1)].normal);

		//check refraction
		float3 refracted = make_float3(0.f);
		float reflectance;
		if(lightPathBuffer[make_uint3(launch_index, i-1)].Ni>0.f && optix::refract(refracted, lightPathBuffer[make_uint3(launch_index, i-1)].In, lightPathBuffer[make_uint3(launch_index, i-1)].normal, lightPathBuffer[make_uint3(launch_index, i-1)].Ni)){
			float cos_theta = dot(lightPathBuffer[make_uint3(launch_index, i-1)].In, lightPathBuffer[make_uint3(launch_index, i-1)].normal);
			if(cos_theta<0.f)
				cos_theta = -cos_theta;
			else
				cos_theta = dot(refracted, lightPathBuffer[make_uint3(launch_index, i-1)].normal);
			float r0 = ((1.f-lightPathBuffer[make_uint3(launch_index, i-1)].Ni)/(1.f+lightPathBuffer[make_uint3(launch_index, i-1)].Ni))*((1.f-lightPathBuffer[make_uint3(launch_index, i-1)].Ni)/(1.f+lightPathBuffer[make_uint3(launch_index, i-1)].Ni));
			reflectance = r0 + (1.f-r0)*powf(1.f-cos_theta, 5.f);

		}
		else reflectance = 1.f;



		float pdiff=(diff_coef.x+diff_coef.y+diff_coef.z)*0.33333333333333333333333333333f;
		float pspec=(spec_coef.x+spec_coef.y+spec_coef.z)*0.33333333333333333333333333333f;
		pspec*=fminf(1.f, optix::dot(-lightPathBuffer[make_uint3(launch_index, i-1)].In, ffnormal)*(lightPathBuffer[make_uint3(launch_index, i-1)].Ns+2.f)/(lightPathBuffer[make_uint3(launch_index, i-1)].Ns+1.f));

		//randomly select the type of contribution
		float r=rnd(seed);
		if(i < MIN_DEPTH || pdiff+pspec>1.f){
			float inv_p = 1.f/(pdiff+pspec);
			pdiff*=inv_p;
			pspec*=inv_p;
		}

		float p_reflect = rnd(seed);

		if(p_reflect<reflectance){
			if(r<pdiff+pspec){
				if(r<pdiff){
					LightPathResult result;
					result.radiance = lightPathBuffer[make_uint3(launch_index, i-1)].radiance * diff_coef/pdiff;
					float3 new_dir;
					optix::cosine_sample_hemisphere(rnd(seed), rnd(seed), new_dir);
					optix::Onb onb(ffnormal);
					onb.inverse_transform(new_dir);
					result.In=new_dir;
					Ray new_ray = optix::make_Ray(lightPathBuffer[make_uint3(launch_index, i-1)].position, lightPathBuffer[make_uint3(launch_index, i)].In, LightPathRay, scene_epsilon, RT_DEFAULT_MAX);
					rtTrace(top_object, new_ray, result);
					lightPathBuffer[make_uint3(launch_index, i)]=result;
				}
				else{
					float u1=rnd(seed);
					float u2=rnd(seed);
					float3 dir;
					dir.x = sqrtf(1-powf(u1, 2.f/(lightPathBuffer[make_uint3(launch_index, i-1)].Ns+1.f)))*cosf(M_2_PIf*u2);
					dir.y = sqrtf(1-powf(u1, 2.f/(lightPathBuffer[make_uint3(launch_index, i-1)].Ns+1.f)))*sinf(M_2_PIf*u2);
					dir.z = powf(u1, 1.f/(lightPathBuffer[make_uint3(launch_index, i-1)].Ns+1.f));
					float3 reflected = optix::reflect(lightPathBuffer[make_uint3(launch_index, i-1)].In, ffnormal);
					optix::Onb onb(reflected);
					onb.inverse_transform(dir);
					LightPathResult result;
					result.In=dir;
					float intensity=optix::dot(dir, ffnormal);

					//verify if sampled direction is above surface
					if(intensity>0.f){
						result.radiance =lightPathBuffer[make_uint3(launch_index, i-1)].radiance * ((lightPathBuffer[make_uint3(launch_index, i-1)].Ns+2.f)/(lightPathBuffer[make_uint3(launch_index, i-1)].Ns+1.f)) * (spec_coef/pspec) * intensity;
						Ray new_ray = optix::make_Ray(lightPathBuffer[make_uint3(launch_index, i-1)].position, lightPathBuffer[make_uint3(launch_index, i)].In, LightPathRay, scene_epsilon, RT_DEFAULT_MAX);
						rtTrace(top_object, new_ray, result);
						lightPathBuffer[make_uint3(launch_index, i)]=result;
					}
					else{
						lightPathBuffer[make_uint3(launch_index, i)].missed=true;
					}
				}
			}
			else{
				lightPathBuffer[make_uint3(launch_index, i)].missed=true;
			}
		}
		else{

			if(r<pdiff+pspec){
				if(r<pdiff){
					LightPathResult result;
					result.radiance = lightPathBuffer[make_uint3(launch_index, i-1)].radiance * diff_coef/pdiff;
					float3 new_dir;
					optix::cosine_sample_hemisphere(rnd(seed), rnd(seed), new_dir);
					optix::Onb onb(-ffnormal);
					onb.inverse_transform(new_dir);
					result.In=new_dir;
					Ray new_ray = optix::make_Ray(lightPathBuffer[make_uint3(launch_index, i-1)].position, lightPathBuffer[make_uint3(launch_index, i)].In, LightPathRay, scene_epsilon, RT_DEFAULT_MAX);
					rtTrace(top_object, new_ray, result);
					lightPathBuffer[make_uint3(launch_index, i)]=result;
				}
				else{
					LightPathResult result;
					float u1=rnd(seed);
					float u2=rnd(seed);
					float3 dir;
					dir.x = sqrtf(1-powf(u1, 2.f/(lightPathBuffer[make_uint3(launch_index, i-1)].Ns+1.f)))*cosf(M_2_PIf*u2);
					dir.y = sqrtf(1-powf(u1, 2.f/(lightPathBuffer[make_uint3(launch_index, i-1)].Ns+1.f)))*sinf(M_2_PIf*u2);
					dir.z = powf(u1, 1.f/(lightPathBuffer[make_uint3(launch_index, i-1)].Ns+1.f));
					optix::Onb onb(refracted);
					onb.inverse_transform(dir);

					result.In=dir;
					float intensity=optix::dot(-dir, ffnormal);

					//verify if sampled direction is above surface
					if(intensity>0.f){
						result.radiance =lightPathBuffer[make_uint3(launch_index, i-1)].radiance * ((lightPathBuffer[make_uint3(launch_index, i-1)].Ns+2.f)/(lightPathBuffer[make_uint3(launch_index, i-1)].Ns+1.f)) * (spec_coef/pspec) * intensity;
						Ray new_ray = optix::make_Ray(lightPathBuffer[make_uint3(launch_index, i-1)].position, lightPathBuffer[make_uint3(launch_index, i)].In, LightPathRay, scene_epsilon, RT_DEFAULT_MAX);
						rtTrace(top_object, new_ray, result);
						lightPathBuffer[make_uint3(launch_index, i)]=result;
					}
					else{
						lightPathBuffer[make_uint3(launch_index, i)].missed=true;
					}
				}
			}
			else{
				lightPathBuffer[make_uint3(launch_index, i)].missed=true;
			}
		}

		i++;
	}

}

RT_PROGRAM void camera(){
	unsigned int seed = tea<16>(launch_dim.x*launch_index.y+launch_index.x, frame);

	float2 inv_screen=1.f/(make_float2(launch_dim)) *2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;
	float2 jitter_scale = inv_screen / sqrt_num_samples;

	int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;

	float4 result = make_float4(0.0f);

	do{
		int x = samples_per_pixel%sqrt_num_samples;
		int y = samples_per_pixel/sqrt_num_samples;
		float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
		float2 d = pixel + jitter*jitter_scale;

		float r = lens_radius * sqrtf(rnd(seed));
		float ang = 2.f * M_PIf * rnd(seed);

		float3 ray_origin = eye + r * ( U * cosf(ang) + V * sinf(ang));
		float3 ray_target = eye + (d.x * U + d.y * V + W) * focal_dist;
		float3 ray_direction = normalize(ray_target - ray_origin);


		PathResult ray_result;
		ray_result.atenuation=make_float4(1.f);
		ray_result.count_emissive=true;
		ray_result.depth=0;
		ray_result.result=make_float4(0.f);
		ray_result.seed=seed;
		ray_result.finished=false;
		ray_result.weight=1.f;

		for(;;){

			Ray ray = make_Ray(ray_origin, ray_direction, PathRay, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, ray_result);

			if(ray_result.finished)
				break;

			ray_origin=ray_result.position;
			ray_direction=ray_result.direction;
			ray_result.depth++;

		}

		result+=ray_result.result;
		seed=ray_result.seed;

	}while(--samples_per_pixel);

	result/=sqrt_num_samples * sqrt_num_samples;


	if(frame>1){
		float a = 1.f/float(frame);
		float b = float(frame-1)*a;
		float4 old_color=output[launch_index];///(make_float4(1.f) - output[launch_index]);
		float4 new_color = a*result+b*old_color;
		output[launch_index]= new_color;///(new_color + make_float4(1.f));
	}
	else{
		output[launch_index]=result;///(result + make_float4(1.f));
	}

}



RT_PROGRAM void exception(){
	//output[launch_index]=make_float4(1.f);
	rtPrintExceptionDetails();
}

RT_PROGRAM void path_miss(){
	current_path_result.finished = true;
}

#include "material.h"


RT_PROGRAM void lightPathTrace(){
	current_light_result.Kd=Kd*tex2D(map_Kd, texCoord.x, texCoord.y);
	current_light_result.Ks=Ks*tex2D(map_Ks, texCoord.x, texCoord.y);
	current_light_result.Ni=Ni;
	current_light_result.Ns=Ns;
	current_light_result.normal=shading_normal;
	current_light_result.position=current_ray.origin + current_ray.direction * t_hit;
	current_light_result.missed=false;
}

RT_PROGRAM void lightPathMiss(){
	current_light_result.missed=true;
}

RT_PROGRAM void lightPathHitLight(){
	current_light_result.missed=true;
}

RT_PROGRAM void glossy_shading(){



	//because we calculate direct lighting in every point of the path,
	//when first diffuse material is hit we stop counting emmisive contributions
	current_path_result.count_emissive=false;
	//calculate diffuse and specular probabilities.
	float4 diff_coef = Kd*tex2D(map_Kd, texCoord.x, texCoord.y);
	float4 spec_coef = Ks*tex2D(map_Ks, texCoord.x, texCoord.y);

	float dc = (diff_coef.x + diff_coef.y + diff_coef.z)*0.33333333333333333333333333333f;
	float ds = (spec_coef.x + spec_coef.y + spec_coef.z)*0.33333333333333333333333333333f;




	float3 position = current_ray.origin + current_ray.direction * t_hit;

	float3 ffnormal = optix::faceforward(shading_normal, -current_ray.direction, shading_normal);

	//check refraction
	float3 refracted = make_float3(0.f);
	float reflectance;
	if(Ni>0 && optix::refract(refracted, current_ray.direction, shading_normal, Ni)){
		float cos_theta = dot(current_ray.direction, shading_normal);
		if(cos_theta<0.f)
			cos_theta = -cos_theta;
		else
			cos_theta = dot(refracted, shading_normal);
		float r0 = ((1.f-Ni)/(1.f+Ni))*((1.f-Ni)/(1.f+Ni));
		reflectance = r0 + (1.f-r0)*powf(1-cos_theta, 5.f);

	}
	else reflectance = 1.f;

	for(int i=0; i<lights.size(); i++){
		//sample light
		float3 center = make_float3(lights[i].pos);

		float3 w = normalize(center-position);
		float3 v = normalize(cross(w, ffnormal));
		float3 u = cross(v, w);

		float u1, u2;
		u1=rnd(current_path_result.seed);
		u2=rnd(current_path_result.seed);

		float cos_a = 1 - u1 + u1 * sqrtf(1-powf(lights[i].pos.w / length(position-center), 2.f));
		float sin_a = sqrtf(1-cos_a*cos_a);
		float phi = 2 * M_PIf * u2;

		float3 dir = u * cosf(phi) * sin_a + v * sinf(phi) * sin_a + w * cos_a;

		float intensity = dot(shading_normal, dir);

		if(intensity>0.f){

			float radius = lights[i].pos.w;

			float3 o = position - center;

			float b = dot(o, dir);
			float c = dot(o, o) - radius * radius;
			float disc = b * b - c;

			float sdisc = sqrtf(disc);
			float root1 = (-b -sdisc);

			Ray shadow_test = make_Ray(position, dir, ShadowRay, scene_epsilon, root1);
			ShadowResult s_res;
			s_res.in_shadow=false;
			rtTrace(top_object, shadow_test, s_res);

			if(!s_res.in_shadow){
				float4 diff_res = diff_coef ;
				/*
				float spec_intensity;
				if(dot(dir, ffnormal)>0.f)
					spec_intensity = fmaxf(dot(dir, reflect(current_ray.direction, ffnormal)), 0.f);
				else{
					if(optix::length(refracted)>0.f){
						spec_intensity = fmaxf(dot(dir, refracted), 0.f);
					}
				}
				spec_intensity = powf(spec_intensity, Ns);
				float4 spec_res = spec_coef * (Ns +2.f)* 0.5f * spec_intensity;*/
				current_path_result.result += ((diff_res) * M_1_PIf * lights[i].color * intensity * current_path_result.atenuation * (1.f - sqrtf(1.f - powf(radius/length(position-center), 2.f))) * 2.f * M_PIf) * current_path_result.weight;
			}
		}
	}

	float weight = (ds/(dc+ds))*.5f + 0.5f;

	if(weight<1.f){
		float4 light_path_contribution=make_float4(0.f);
		for(int i=0 ; i<LIGHT_PATH_LENGTH; i++){
			uint3 lindex = make_uint3(launch_index, i);
			if(lightPathBuffer[lindex].missed) break;

			float3 dir = lightPathBuffer[lindex].position - position;
			float tdist = length(dir);
			dir = normalize(dir);

			Ray shadow_r = optix::make_Ray(position, dir, ShadowRay, scene_epsilon, tdist);
			ShadowResult sres;
			sres.in_shadow=false;

			rtTrace(top_object, shadow_r, sres);
			if(!sres.in_shadow){


				//check refraction
				float3 out_refracted = make_float3(0.f);
				float out_reflectance;
				if(lightPathBuffer[lindex].Ni>0.f && optix::refract(out_refracted, lightPathBuffer[lindex].In, lightPathBuffer[lindex].normal, lightPathBuffer[lindex].Ni)){
					float cos_theta = dot(lightPathBuffer[lindex].In, lightPathBuffer[lindex].normal);
					if(cos_theta<0.f)
						cos_theta = -cos_theta;
					else
						cos_theta = dot(out_refracted, lightPathBuffer[lindex].normal);
					float r0 = ((1.f-lightPathBuffer[lindex].Ni)/(1.f+lightPathBuffer[lindex].Ni))*((1.f-lightPathBuffer[lindex].Ni)/(1.f+lightPathBuffer[lindex].Ni));

					out_reflectance = r0 + (1.f-r0)*powf(1.f-cos_theta, 5.f);

				}
				else out_reflectance = 1.f;

				float3 spec_dir;
				float spec_intensity;
				if(dot(dir, lightPathBuffer[lindex].In)>0.f){
					spec_dir=optix::reflect(lightPathBuffer[lindex].In, lightPathBuffer[lindex].normal);
					spec_intensity=fmaxf(0.f,powf(dot(spec_dir, -dir), lightPathBuffer[lindex].Ns));
				}
				else{
					if(optix::length(out_refracted)>0.f){
						spec_intensity=fmaxf(0.f,powf(dot(out_refracted, -dir), lightPathBuffer[lindex].Ns));
					}
					else{
						spec_intensity=0.f;
					}
				}
				float out_intensity = abs(dot(lightPathBuffer[lindex].In, lightPathBuffer[lindex].normal));

				float4 out_rad_diff = lightPathBuffer[lindex].radiance * lightPathBuffer[lindex].Kd * M_1_PIf;;
				float4 out_rad_spec = lightPathBuffer[lindex].radiance * lightPathBuffer[lindex].Ks * spec_intensity * (lightPathBuffer[lindex].Ns + 2.f) * 0.5 * M_1_PIf;

				float4 out_rad = (out_rad_diff+out_rad_spec) * out_intensity;
				if(dot(dir, lightPathBuffer[lindex].In)>0.f) out_rad *= out_reflectance;
				else out_rad *= (1.f - out_reflectance);


				float in_intensity = abs(dot(dir, ffnormal));


				float in_spec_intensity;
				if(dot(current_ray.direction, dir)<0.f){
					in_spec_intensity = fmaxf(0.f, powf(dot(optix::reflect(current_ray.direction, ffnormal),dir), Ns));
				}
				else{

					if(optix::length(refracted)>0.f){
						in_spec_intensity = fmaxf(0.f, powf(dot(refracted, dir), Ns));
					}
					else{
						in_spec_intensity=0.f;
					}
				}

				float4 in_rad_diff = out_rad * diff_coef * M_1_PIf;
				float4 in_rad_spec = out_rad * spec_coef * in_spec_intensity * (Ns + 2) * 0.5 * M_1_PIf;

				float4 in_rad = (in_rad_diff+in_rad_spec) * in_intensity * fminf(abs(dot(dir, lightPathBuffer[lindex].normal)) / (tdist*tdist), 1.f);
				if(dot(current_ray.direction, dir)<0.f) in_rad *= reflectance;
				else in_rad *= 1.f - reflectance;

				current_path_result.result+=in_rad * current_path_result.weight * (1.f-weight);
				light_path_contribution+=in_rad * current_path_result.weight * (1.f-weight);
			}
		}
		if(light_path_contribution.x+light_path_contribution.y+light_path_contribution.z==0.f){
			weight=1.f;
		}
	}

	if(weight==0.f) {
		current_path_result.finished=true;
		return;
	}

	current_path_result.weight*=weight;

	float3 pkd = make_float3(diff_coef*current_path_result.atenuation);
	float3 pks = make_float3(spec_coef*current_path_result.atenuation);

	float pdiff=(pkd.x+pkd.y+pkd.z)*0.33333333333333333333333333333f;
	float pspec=(pks.x+pks.y+pks.z)*0.33333333333333333333333333333f;
	pspec*=fminf(1.f, optix::dot(-current_ray.direction, ffnormal)*(Ns+2.f)/(Ns+1.f));

	//randomly select the type of contribution
	float r=rnd(current_path_result.seed);
	if(current_path_result.depth < MIN_DEPTH || pdiff+pspec>1.f){
		float inv_p = 1.f/(pdiff+pspec);
		pdiff*=inv_p;
		pspec*=inv_p;
	}

	float preflect = rnd(current_path_result.seed);
	if(preflect < reflectance){

		if(r<pdiff+pspec){
			//select diffuse sample
			if(r<pdiff){

				float u1=rnd(current_path_result.seed);
				float u2=rnd(current_path_result.seed);
				float3 dir;
				optix::cosine_sample_hemisphere(u1, u2, dir);
				optix::Onb onb(ffnormal);
				onb.inverse_transform(dir);

				current_path_result.atenuation *= diff_coef/pdiff;
				current_path_result.direction = dir;
				current_path_result.position = position;

			}
			//select specular sample

			else {
				current_path_result.count_emissive=true;
				float u1=rnd(current_path_result.seed);
				float u2=rnd(current_path_result.seed);
				float3 dir;
				dir.x = sqrtf(1-powf(u1, 2.f/(Ns+1.f)))*cosf(M_2_PIf*u2);
				dir.y = sqrtf(1-powf(u1, 2.f/(Ns+1.f)))*sinf(M_2_PIf*u2);
				dir.z = powf(u1, 1.f/(Ns+1.f));
				optix::Onb onb(optix::reflect(current_ray.direction, ffnormal));
				onb.inverse_transform(dir);

				float intensity=optix::dot(dir, ffnormal);
				//verify if sampled direction is above surface
				if(intensity>0.f){
					current_path_result.atenuation*= ((Ns+2.f)/(Ns+1.f)) * (spec_coef/pspec) * intensity;
					current_path_result.direction=dir;
					current_path_result.position = position;
				}
				else{
					current_path_result.finished=true;
				}
			}
		}
		//consider that photon is absorbed and finish path
		else{
			current_path_result.finished=true;
		}
	}

	else{

		if(r<pdiff+pspec){
			//select diffuse sample
			if(r<pdiff){

				float u1=rnd(current_path_result.seed);
				float u2=rnd(current_path_result.seed);
				float3 dir;
				optix::cosine_sample_hemisphere(u1, u2, dir);
				optix::Onb onb(-ffnormal);
				onb.inverse_transform(dir);

				current_path_result.atenuation *= diff_coef/pdiff;
				current_path_result.direction = dir;
				current_path_result.position = position;

			}
			//select specular sample

			else {
				current_path_result.count_emissive=true;
				float u1=rnd(current_path_result.seed);
				float u2=rnd(current_path_result.seed);
				float3 dir;
				dir.x = sqrtf(1-powf(u1, 2.f/(Ns+1.f)))*cosf(M_2_PIf*u2);
				dir.y = sqrtf(1-powf(u1, 2.f/(Ns+1.f)))*sinf(M_2_PIf*u2);
				dir.z = powf(u1, 1.f/(Ns+1.f));
				optix::Onb onb(refracted);
				onb.inverse_transform(dir);

				float intensity;
				intensity=optix::dot(dir, -ffnormal);


				//verify if sampled direction is above surface
				if(intensity>0.f){
					current_path_result.atenuation*= ((Ns+2.f)/(Ns+1.f)) * (spec_coef/pspec) * intensity;
					current_path_result.direction=dir;
					current_path_result.position = position;
				}
				else{
					current_path_result.finished=true;
				}
			}
		}
		//consider that photon is absorbed and finish path
		else{
			current_path_result.finished=true;
		}
	}
}

RT_PROGRAM void path_ignore_alpha(){
	float4 color=Kd*tex2D(map_Kd, texCoord.x, texCoord.y);
	if(color.w == 0.f) rtIgnoreIntersection();
}


RT_PROGRAM void shadow_probe(){
	float4 color=Kd*tex2D(map_Kd, texCoord.x, texCoord.y);
	if(color.w == 0.f) rtIgnoreIntersection();
	else{
		current_shadow_result.in_shadow=true;
		rtTerminateRay();
	}
}

RT_PROGRAM void light_shading(){

	if(current_path_result.count_emissive) current_path_result.result += light_color * current_path_result.atenuation*current_path_result.weight;
	current_path_result.finished=true;
}

RT_PROGRAM void shadow_probe_light(){
	current_shadow_result.in_shadow=true;
	rtTerminateRay();
}


