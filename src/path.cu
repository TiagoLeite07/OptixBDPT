#include "hip/hip_runtime.h"
#include <optix_world.h>

struct PathResult{
	float4 result;
	float4 atenuation;
	float3 position;
	float3 direction;
	unsigned int depth;
	unsigned int seed;
	bool count_emissive;
	bool finished;
};

rtDeclareVariable(PathResult, current_path_result, rtPayload, );

__device__ __inline__ void calc_direct_light(){

}

#include "material.h"


RT_PROGRAM void closest_hit(){
	//because we calculate direct lighting in every point of the path,
	//when first diffuse material is hit we stop counting emmisive contributions
	current_path_result.count_emissive=false;
	//calculate diffuse and specular probabilities.
	float pdiff=(Kd.x+Kd.y+Kd.z)*0.33333333333333333333333333333f;
	float pspec=(Ks.x+Ks.y+Ks.z)*0.33333333333333333333333333333f;
	pspec*=fminf(1.f, optix::dot(current_ray.direction, shading_normal)*(Ns+2.f)/(Ns+1.f));

	int * a = (int*) malloc(sizeof(int));

	//randomly select the type of contribution
	float r=rnd(current_path_result.seed);
	if(r<pdiff+pspec){
		//select diffuse sample
		if(r<pdiff){
			float u1=rnd(current_path_result.seed);
			float u2=rnd(current_path_result.seed);
			float3 dir;
			optix::cosine_sample_hemisphere(u1, u2, dir);
			optix::Onb onb(shading_normal);
			onb.inverse_transform(dir);

			current_path_result.atenuation *= Kd/pdiff;
			current_path_result.direction = dir;

		}
		//select specular sample
		else {
			float u1=rnd(current_path_result.seed);
			float u2=rnd(current_path_result.seed);
			float3 dir;
			dir.x = sqrtf(1-powf(u1, 2.f/(Ns+1.f)))*cosf(M_2_PIf*u2);
			dir.y = sqrtf(1-powf(u1, 2.f/(Ns+1.f)))*sinf(M_2_PIf*u2);
			dir.z = powf(u1, 1.f/(Ns+1.f));
			optix::Onb onb(optix::reflect(current_ray.direction, shading_normal));
			onb.inverse_transform(dir);

			float intensity=optix::dot(dir, shading_normal);
			//verify if sampled direction is above surface
			if(intensity>0.f){
				current_path_result.atenuation*= ((Ns+2.f)/(Ns+1.f)) * (Ks/pspec) * optix::dot(dir, shading_normal);
				current_path_result.direction=dir;
			}
			else{
				current_path_result.finished=true;
			}
		}
	}
	//consider that photon is absorbed and finish path
	else{
		current_path_result.finished=true;
	}

}
